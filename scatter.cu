#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <stdio.h>
#include <iostream>
#include <ctime>
#include <random>

#include "GpuTimer.h"

#include "tclap/CmdLine.h"

#define EPS 0.01
#define BLOCK_WIDTH 16

#define LATTICE_DATA_TYPE float

__global__ void DCSKernel(LATTICE_DATA_TYPE *slice, const float *atomXs, const float *atomYs, const float *atomZs, const float *charges, const unsigned short int z, const unsigned int numOfAtoms, const unsigned short int latticeX, const unsigned short int latticeY, const LATTICE_DATA_TYPE latticeGridSpacing)
{
	unsigned int atomIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if (atomIdx < numOfAtoms) {
		float atomX = atomXs[atomIdx];
		float atomY = atomYs[atomIdx];
		float atomZ = atomZs[atomIdx];
		float charge = charges[atomIdx];

		const unsigned int latticeSliceGridSize = latticeX * latticeY;
		unsigned int sliceYOffset;
		unsigned long int sliceIdx;

		LATTICE_DATA_TYPE dx, dy, dz, dx2, dy2, dz2, dy2dz2, distance;
		LATTICE_DATA_TYPE potential;

		dz = atomZ - z * latticeGridSpacing;
		dz2 = dz * dz;
		for (unsigned short int y = 0; y < latticeY; y++) {
			sliceYOffset = latticeX * y;
			dy = atomY - y * latticeGridSpacing;
			dy2 = dy * dy;
			dy2dz2 = dy2 + dz2;
			for (unsigned short int x = 0; x < latticeX; x++) {
				dx = atomX - x * latticeGridSpacing;
				dx2 = dx * dx;

				distance = sqrt(dx2 + dy2dz2);
				potential = charge / distance;

				sliceIdx = sliceYOffset + x;
				atomicAdd(&slice[sliceIdx], potential);
			}
		}
	}
}

void CPU(LATTICE_DATA_TYPE *lattice, const float *atomXs, const float *atomYs, const float *atomZs, const float *charges, const unsigned short int z, const unsigned int numOfAtoms, const unsigned short int latticeX, const unsigned short int latticeY, const LATTICE_DATA_TYPE latticeGridSpacing)
{
	float atomX, atomY, atomZ, charge;
	
	const unsigned int latticeSliceGridSize = latticeX * latticeY;
	unsigned long int latticeZOffset;
	unsigned int latticeYOffset;
	unsigned long int latticeOffset;
	unsigned long int latticeIdx;

	LATTICE_DATA_TYPE dx, dy, dz, dx2, dy2, dz2, dy2dz2, distance;
	LATTICE_DATA_TYPE potential;

	for (unsigned int atomIdx = 0; atomIdx < numOfAtoms; atomIdx++) {
		atomX = atomXs[atomIdx];
		atomY = atomYs[atomIdx];
		atomZ = atomZs[atomIdx];
		charge = charges[atomIdx];
		latticeZOffset = latticeSliceGridSize * z;
		dz = atomZ - z * latticeGridSpacing;
		dz2 = dz * dz;
		for (unsigned short int y = 0; y < latticeY; y++) {
			latticeYOffset = latticeX * y;
			latticeOffset = latticeZOffset + latticeYOffset;
			dy = atomY - y * latticeGridSpacing;
			dy2 = dy * dy;
			dy2dz2 = dy2 + dz2;
			for (unsigned short int x = 0; x < latticeX; x++) {
				dx = atomX - x * latticeGridSpacing;
				dx2 = dx * dx;
				distance = sqrt(dx2 + dy2dz2);
				potential = charge / distance;
				latticeIdx = latticeOffset + x;
				lattice[latticeIdx] += potential;
			}
		}
	}
}

int main(int argc, char *argv[])
{
	double latticeW;
	double latticeH;
	double latticeD;
	double latticeGridSpacing;
	unsigned int numOfAtoms;
	float maxCharge;
	uint8_t numOfStreams;

	try {
		TCLAP::CmdLine cmd("Runs the Direct Couloumb Summation algorithm on the CPU & GPU (CUDA).", ' ', "1.0");
		TCLAP::ValueArg<double> latticeWArg("x", "width", "Lattice width", true, 1.0f, "double");
		TCLAP::ValueArg<double> latticeHArg("y", "height", "Lattice height", true, 1.0f, "double");
		TCLAP::ValueArg<double> latticeDArg("z", "depth", "Lattice depth", true, 1.0f, "double");
		TCLAP::ValueArg<double> latticeGridSpacingArg("g", "spacing", "Lattice grid spacing", true, 0.1f, "double");
		TCLAP::ValueArg<unsigned int> numOfAtomsArg("a", "atoms", "Number of atoms", true, 1, "int");
		TCLAP::ValueArg<double> maxChargeArg("c", "charge", "Maximum charge", true, 1.0f, "double");
		TCLAP::ValueArg<unsigned int> numOfStreamsArg("n", "streams", "Number of CUDA streams", false, 2, "int");

		cmd.add(numOfStreamsArg);
		cmd.add(maxChargeArg);
		cmd.add(numOfAtomsArg);
		cmd.add(latticeGridSpacingArg);
		cmd.add(latticeDArg);
		cmd.add(latticeHArg);
		cmd.add(latticeWArg);

		cmd.parse(argc, argv);

		latticeW = latticeWArg.getValue();
		latticeH = latticeHArg.getValue();
		latticeD = latticeDArg.getValue();
		latticeGridSpacing = latticeGridSpacingArg.getValue();
		numOfAtoms = numOfAtomsArg.getValue();
		maxCharge = maxChargeArg.getValue();
		numOfStreams = numOfStreamsArg.getValue();
	}
	catch (TCLAP::ArgException &e) {
		fprintf(stderr, "Error in argument(s): %s\n", e.what());
		return 1;
	}

	const unsigned short int latticeX = floor(latticeW / latticeGridSpacing) + 1;
	const unsigned short int latticeY = floor(latticeH / latticeGridSpacing) + 1;
	const unsigned short int latticeZ = floor(latticeD / latticeGridSpacing) + 1;
	const unsigned long int sliceGridSize = latticeX * latticeY;
	const unsigned long int latticeGridSize = sliceGridSize * latticeZ;

	float *h_AtomX, *h_AtomY, *h_AtomZ;
	float *h_Charge;
	float *d_AtomX, *d_AtomY, *d_AtomZ;
	float *d_Charge;

	LATTICE_DATA_TYPE *latticeCPU;
	LATTICE_DATA_TYPE *h_LatticeDCS;
	LATTICE_DATA_TYPE **d_SliceDCS;

	hipError_t cudaStatus;

	std::default_random_engine generator;
	std::uniform_real_distribution<float> latticeXDistribution(0, latticeX - 1);
	std::uniform_real_distribution<float> latticeYDistribution(0, latticeY - 1);
	std::uniform_real_distribution<float> latticeZDistribution(0, latticeZ - 1);
	std::uniform_real_distribution<float> chargeDistribution(0, maxCharge);

	uint8_t numOfRemainingLaunches;
	uint8_t streamIdx;
	hipStream_t *stream;
	unsigned long h_LatticeDCSOffset;

	clock_t mallocClock;
	double mallocDuration;
	GpuTimer cudaMallocTimer;
	GpuTimer cudaMemcpyHostDeviceTimer;
	clock_t randomGenerationClock;
	double randomGenerationDuration;
	GpuTimer DCSKernelTimer;
	clock_t CPUClock;
	double CPUDuration;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	mallocClock = clock();

	stream = (hipStream_t*)malloc(numOfStreams * sizeof(hipStream_t));

	hipHostAlloc((void**)&h_AtomX, numOfAtoms * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&h_AtomY, numOfAtoms * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&h_AtomZ, numOfAtoms * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&h_Charge, numOfAtoms * sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void**)&h_LatticeDCS, latticeGridSize * sizeof(LATTICE_DATA_TYPE), hipHostMallocDefault);
	hipHostAlloc((void**)&d_SliceDCS, numOfStreams * sizeof(LATTICE_DATA_TYPE*), hipHostMallocDefault);

	latticeCPU = (LATTICE_DATA_TYPE*)malloc(latticeGridSize * sizeof(LATTICE_DATA_TYPE));
	memset(latticeCPU, 0, latticeGridSize * sizeof(LATTICE_DATA_TYPE));

	mallocDuration = (clock() - mallocClock) / (double)CLOCKS_PER_SEC;
	printf("Memory allocation (host): %f ms\n", mallocDuration * 1000);

	cudaMallocTimer.Start();

	cudaStatus = hipMalloc((void**)&d_AtomX, numOfAtoms * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc (atomX) failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_AtomY, numOfAtoms * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc (atomY) failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_AtomZ, numOfAtoms * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc (atomZ) failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_Charge, numOfAtoms * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc (charge) failed!");
		goto Error;	
	}

	for (streamIdx = 0; streamIdx < numOfStreams; streamIdx++) {
		cudaStatus = hipMalloc((void**)&d_SliceDCS[streamIdx], sliceGridSize * sizeof(LATTICE_DATA_TYPE));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc (DCS, slice[%i]) failed!", streamIdx);
			goto Error;
		}
	}

	cudaMallocTimer.Stop();
	printf("Memory allocation (device): %f ms\n", cudaMallocTimer.Elapsed());

	randomGenerationClock = clock();
	for (unsigned int i = 0; i < numOfAtoms; i++) {
		h_AtomX[i] = latticeXDistribution(generator);
		h_AtomY[i] = latticeYDistribution(generator);
		h_AtomZ[i] = latticeZDistribution(generator);
		h_Charge[i] = chargeDistribution(generator);
	}
	randomGenerationDuration = (clock() - randomGenerationClock) / (double)CLOCKS_PER_SEC;
	printf("Random generation (CPU): %f ms\n", randomGenerationDuration * 1000);

	cudaMemcpyHostDeviceTimer.Start();

	cudaStatus = hipMemcpy(d_AtomX, h_AtomX, numOfAtoms * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy (atomX, host -> device) failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_AtomY, h_AtomY, numOfAtoms * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy (atomY, host -> device) failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_AtomZ, h_AtomZ, numOfAtoms * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy (atomZ, host -> device) failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_Charge, h_Charge, numOfAtoms * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy (charge, host -> device) failed!");
		goto Error;
	}

	cudaMemcpyHostDeviceTimer.Stop();
	printf("Memory copy (host -> device): %f ms\n", cudaMemcpyHostDeviceTimer.Elapsed());

	dim3 dimBlockDCS(BLOCK_WIDTH * BLOCK_WIDTH, 1, 1);
	dim3 dimGridDCS((numOfAtoms - 1) / dimBlockDCS.x + 1, 1, 1);

	for (uint8_t streamIdx = 0; streamIdx < numOfStreams; streamIdx++) {
		hipStreamCreate(&stream[streamIdx]);
	}

	DCSKernelTimer.Start();
	h_LatticeDCSOffset = 0;
	if (latticeZ > 1) {
		for (unsigned short int z = 0; z < latticeZ; z += numOfStreams) {
			for (streamIdx = 0; streamIdx < numOfStreams; streamIdx++) {
				hipMemsetAsync(d_SliceDCS[streamIdx], 0, sliceGridSize * sizeof(LATTICE_DATA_TYPE), stream[streamIdx]);
			}
			
			for (streamIdx = 0; streamIdx < numOfStreams; streamIdx++) {
				DCSKernel << <dimGridDCS, dimBlockDCS, 0, stream[streamIdx] >> >(d_SliceDCS[streamIdx], d_AtomX, d_AtomY, d_AtomZ, d_Charge, z + streamIdx, numOfAtoms, latticeX, latticeY, latticeGridSpacing);
			}

			for (streamIdx = 0; streamIdx < numOfStreams; streamIdx++) {
				h_LatticeDCSOffset = (z + streamIdx) * sliceGridSize;
				hipMemcpyAsync(h_LatticeDCS + h_LatticeDCSOffset, d_SliceDCS[streamIdx], sliceGridSize * sizeof(LATTICE_DATA_TYPE), hipMemcpyDeviceToHost, stream[streamIdx]);
			}
		}
	}
	
	numOfRemainingLaunches = latticeZ % numOfStreams;
	if (numOfRemainingLaunches != 0) {
		unsigned short int z = (latticeZ - numOfStreams);
		for (streamIdx = 0; streamIdx < numOfRemainingLaunches; streamIdx++) {
			hipMemsetAsync(d_SliceDCS[streamIdx], 0, sliceGridSize * sizeof(LATTICE_DATA_TYPE), stream[streamIdx]);
		}

		for (streamIdx = 0; streamIdx < numOfRemainingLaunches; streamIdx++) {
			DCSKernel << <dimGridDCS, dimBlockDCS, 0, stream[streamIdx] >> >(d_SliceDCS[streamIdx], d_AtomX, d_AtomY, d_AtomZ, d_Charge, z + streamIdx, numOfAtoms, latticeX, latticeY, latticeGridSpacing);
		}

		for (streamIdx = 0; streamIdx < numOfRemainingLaunches; streamIdx++) {
			h_LatticeDCSOffset = (z + streamIdx) * sliceGridSize;
			hipMemcpyAsync(h_LatticeDCS + h_LatticeDCSOffset, d_SliceDCS[streamIdx], sliceGridSize * sizeof(LATTICE_DATA_TYPE), hipMemcpyDeviceToHost, stream[streamIdx]);
		}
	}

	for (streamIdx = 0; streamIdx < numOfStreams; streamIdx++) {
		hipStreamSynchronize(stream[streamIdx]);
	}

	DCSKernelTimer.Stop();

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "DCSKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching DCSKernel!\n", cudaStatus);
		goto Error;
	}

	printf("DCSKernel duration: %f ms\n", DCSKernelTimer.Elapsed());

	//CPU
	CPUClock = clock();
	memset(latticeCPU, 0, latticeGridSize * sizeof(LATTICE_DATA_TYPE));
	for (unsigned short int z = 0; z < latticeZ; z++) {
		CPU(latticeCPU, h_AtomX, h_AtomY, h_AtomZ, h_Charge, z, numOfAtoms, latticeX, latticeY, latticeGridSpacing);
	}
	CPUDuration = (clock() - CPUClock) / (double)CLOCKS_PER_SEC;
	printf("CPU duration: %f ms\n", CPUDuration * 1000);

	printf("DCS verification started.\n");
	for (unsigned int i = 0; i < latticeGridSize; i++) {
		if (abs(latticeCPU[i] - h_LatticeDCS[i]) > EPS) {
			fprintf(stderr, "DCS Verification failed at element %i! latticeCPU[%i] = %f, latticeDCS[%i] = %f\n", i, i, latticeCPU[i], i, h_LatticeDCS[i]);
			return 1;
		}
	}
	printf("DCS verification PASSED.\n");

Error:
	free(h_AtomX);
	free(h_AtomY);
	free(h_AtomZ);
	free(latticeCPU);
	free(h_LatticeDCS);
	
	hipFree(d_AtomX);
	hipFree(d_AtomY);
	hipFree(d_AtomZ);
	for (uint8_t streamIdx = 0; streamIdx < numOfStreams; streamIdx++) {
		hipFree(d_SliceDCS[streamIdx]);
	}

    hipError_t cudaStatusReset = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	return cudaStatus;
}